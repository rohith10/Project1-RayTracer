#include "hip/hip_runtime.h"
// CIS565 CUDA Raytracer: A parallel raytracer for Patrick Cozzi's CIS565: GPU Computing at the University of Pennsylvania
// Written by Yining Karl Li, Copyright (c) 2012 University of Pennsylvania
// This file includes code from:
//       Rob Farber for CUDA-GL interop, from CUDA Supercomputing For The Masses: http://www.drdobbs.com/architecture-and-design/cuda-supercomputing-for-the-masses-part/222600097
//       Peter Kutz and Yining Karl Li's GPU Pathtracer: http://gpupathtracer.blogspot.com/
//       Yining Karl Li's TAKUA Render, a massively parallel pathtracing renderer: http://www.yiningkarlli.com

#include <thrust/device_vector.h>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <cmath>
#include "sceneStructs.h"
#include "glm/glm.hpp"
#include "utilities.h"
#include "raytraceKernel.h"
#include "intersections.h"
#include "interactions.h"
//#include <vector>

#if CUDA_VERSION >= 5000
    #include <hip/hip_vector_types.h>
#else
    #include <cutil_math.h>
#endif

const glm::vec3 bgColour = glm::vec3 (0.55, 0.25, 0);

void checkCUDAError(const char *msg) {
  hipError_t err = hipGetLastError();
  if( hipSuccess != err) {
    fprintf(stderr, "Cuda error: %s: %s.\n", msg, hipGetErrorString( err) ); 
    exit(EXIT_FAILURE); 
  }
} 

//Sets up the projection half vectors.
void	setupProjection (projectionInfo &ProjectionParams, glm::vec3 eye, glm::vec3 view, glm::vec3 up, glm::vec2 fov)
{
	//Set up the projection variables:
	float	degToRad = 3.1415926 / 180.0;
	float	radToDeg = 1.0 / degToRad;

	ProjectionParams.centreProj = eye+view;
	glm::vec3	eyeToProjCentre = ProjectionParams.centreProj - eye;
	glm::vec3	A = glm::cross (ProjectionParams.centreProj, up);
	glm::vec3	B = glm::cross (A, ProjectionParams.centreProj);
	float		lenEyeToProjCentre = glm::length (eyeToProjCentre);
	
	ProjectionParams.halfVecH = glm::normalize (A) * lenEyeToProjCentre * (float)tan ((fov.x*degToRad));
	ProjectionParams.halfVecV = glm::normalize (B) * lenEyeToProjCentre * (float)tan ((fov.y*degToRad));
}

// Reflects the incidentRay around the normal.
__host__ __device__ glm::vec3 reflectRay (glm::vec3 incidentRay, glm::vec3 normal)
{
	glm::vec3 reflectedRay = incidentRay - (2.0f*glm::dot (incidentRay, normal))*normal;
	return reflectedRay;
}

//LOOK: This function demonstrates how to use thrust for random number generation on the GPU!
//Function that generates static.
__host__ __device__ glm::vec3 generateRandomNumberFromThread(glm::vec2 resolution, float time, int x, int y){
  int index = x + (y * resolution.x);
   
  thrust::default_random_engine rng(hash(index*time));
  thrust::uniform_real_distribution<float> u01(0,1);

  return glm::vec3((float) u01(rng), (float) u01(rng), (float) u01(rng));
}

//TODO: IMPLEMENT THIS FUNCTION
//Function that does the initial raycast from the camera
__host__ __device__ ray raycastFromCameraKernel(glm::vec2 resolution, float time, int x, int y, glm::vec3 eye, glm::vec3 view, glm::vec3 up, glm::vec2 fov, glm::vec3 centreProj,
													glm::vec3	halfVecH, glm::vec3 halfVecV)
{
  ray r;
  r.origin = eye;
  r.direction = glm::vec3(0,0,-1);

  float normDeviceX = (float)x / (resolution.x-1);
  float normDeviceY = 1 - ((float)y / (resolution.y-1));

  glm::vec3 P = centreProj + (2*normDeviceX - 1)*halfVecH + (2*normDeviceY - 1)*halfVecV;
  r.direction = glm::normalize (P - r.origin);

  return r;
}

//Kernel that blacks out a given image buffer
__global__ void clearImage(glm::vec2 resolution, glm::vec3* image){
    int x = (blockIdx.x * blockDim.x) + threadIdx.x;
    int y = (blockIdx.y * blockDim.y) + threadIdx.y;
    int index = x + (y * resolution.x);
    if(x<=resolution.x && y<=resolution.y){
      image[index] = glm::vec3(0,0,0);
    }
}

//Kernel that writes the image to the OpenGL PBO directly.
__global__ void sendImageToPBO(uchar4* PBOpos, glm::vec2 resolution, glm::vec3* image){
  
  int x = (blockIdx.x * blockDim.x) + threadIdx.x;
  int y = (blockIdx.y * blockDim.y) + threadIdx.y;
  int index = x + (y * resolution.x);
  
  if(x<=resolution.x && y<=resolution.y){

      glm::vec3 color;
      color.x = image[index].x*255.0;
      color.y = image[index].y*255.0;
      color.z = image[index].z*255.0;

      if(color.x>255){
        color.x = 255;
      }

      if(color.y>255){
        color.y = 255;
      }

      if(color.z>255){
        color.z = 255;
      }
      
      // Each thread writes one pixel location in the texture (textel)
      PBOpos[index].w = 0;
      PBOpos[index].x = color.x;
      PBOpos[index].y = color.y;
      PBOpos[index].z = color.z;
  }
}

__device__ interceptInfo getIntercept (staticGeom * geoms, sceneInfo objectCountInfo, ray castRay, material* textureArray)
{
	glm::vec3 intrPoint = glm::vec3 (0, 0, 0);
	glm::vec3 intrNormal = glm::vec3 (0, 0, 0);

	float interceptValue = -32767;

	material newMaterial;
	newMaterial.color = glm::vec3 (0,0,0);
	newMaterial.specularExponent = 1.0;
	newMaterial.hasReflective = 0.0;
	newMaterial.hasRefractive = 0.0;

	interceptInfo theRightIntercept;					// Stores the lowest intercept.
	theRightIntercept.interceptVal = interceptValue;	// Initially, it is empty/invalid
	theRightIntercept.intrNormal = intrNormal;			// Intially, Normal - 0,0,0
	theRightIntercept.intrMaterial = newMaterial;

	float min = 1e6;
	// Two different loops to intersect ray with cubes and spheres.
	for (int i = 0; i < objectCountInfo.nCubes; ++i)
	{
		staticGeom currentGeom = geoms [i];

		interceptValue = boxIntersectionTest(currentGeom, castRay, intrPoint, intrNormal);
		if (interceptValue > 0)
		{
			if (interceptValue < min)
			{
				min = interceptValue;

				theRightIntercept.interceptVal = min;
				theRightIntercept.intrNormal = intrNormal;
				theRightIntercept.intrMaterial = textureArray [currentGeom.materialid];
			}
		}
	}

	for (int i = objectCountInfo.nCubes; i <= (objectCountInfo.nCubes+objectCountInfo.nSpheres); ++i)
	{
		staticGeom currentGeom = geoms [i];

		interceptValue = sphereIntersectionTest(currentGeom, castRay, intrPoint, intrNormal);
		if (interceptValue > 0)
		{
			if (interceptValue < min)
			{
				min = interceptValue;

				theRightIntercept.interceptVal = min;
				theRightIntercept.intrNormal = intrNormal;
				theRightIntercept.intrMaterial = textureArray [currentGeom.materialid];
			}
		}
	}

	return theRightIntercept;
}

__device__ glm::vec3 calcShade (interceptInfo theRightIntercept, glm::vec3 lightVec, glm::vec3 eye, ray castRay, material* textureArray, float ka, float ks, float kd, glm::vec3 lightCol)
{
	glm::vec3 shadedColour = glm::vec3 (0,0,0);
	if (theRightIntercept.interceptVal > 0)
	{
		// Ambient shading
		shadedColour = ka * theRightIntercept.intrMaterial.color;

		// Diffuse shading
		glm::vec3 intrPoint = castRay.origin + theRightIntercept.interceptVal*castRay.direction;
//		glm::vec3 lightVec = glm::normalize (lightPos - intrPoint);	// Now it stores the vector pointing toward the light from the intersection point.
		glm::vec3 intrNormal = glm::normalize (eye - intrPoint); // Refurbish intrNormal for use as the view vector.
		float interceptValue = max (glm::dot (theRightIntercept.intrNormal, lightVec), (float)0); // interceptValue is reused to compute dot product.
		intrPoint = (theRightIntercept.intrMaterial.color * kd * interceptValue);			// Reuse intrPoint to store partial product (kdId) of the diffuse shading computation.
		shadedColour += multiplyVV (/*textureArray [light.materialid].color*/lightCol, intrPoint);		

		// Specular shading	-- TODO: Diffuse surfaces need not be shaded specular - maybe this is why diffuse surfaces are being seen as flat shaded!
		lightVec = glm::normalize (reflectRay (-lightVec, theRightIntercept.intrNormal)); // Reuse lightVec for storing the reflection of light ray around the normal.
		interceptValue = max (glm::dot (lightVec, intrNormal), (float)0);				// Reuse interceptValue for computing dot pdt of specular.
		shadedColour += (/*textureArray [light.materialid].color*/lightCol * ks * pow (interceptValue, theRightIntercept.intrMaterial.specularExponent));
	}
	
	return	shadedColour;
}

//TODO: Done!
//Core raytracer kernel
__global__ void raytraceRay(glm::vec2 resolution, float time, cameraData cam, int rayDepth, glm::vec3* colors,
                            staticGeom* geoms, sceneInfo objectCountInfo, material* textureArray, projectionInfo ProjectionParams)
{
  __shared__ staticGeom light;
  __shared__ float ks;
  __shared__ float ka;
  __shared__ float kd;
  __shared__ glm::vec3 lightPos;
  __shared__ glm::vec3 lightCol;
  __shared__ float nLights;

  if ((threadIdx.x == 0) && (threadIdx.y == 0))
  {
	  ks = 0.3;
	  ka = 0.2;
	  kd = 1-ks-ka;
	  nLights = 9;
	  light = geoms [0];
	  lightPos = /*multiplyMV (light.transform, */glm::vec3 (0, -0.6, 0)/*)*/;
	  lightCol = (textureArray [light.materialid].color/* * textureArray [light.materialid].emittance*/);
  }
  __syncthreads ();

  int x = (blockIdx.x * blockDim.x) + threadIdx.x;
  int y = (blockIdx.y * blockDim.y) + threadIdx.y;
  int index = x + (y * resolution.x);

  glm::vec3 shadedColour = glm::vec3 (0);

  if((x<=resolution.x && y<=resolution.y))
  {
    ray castRay = raycastFromCameraKernel(resolution, time, x, y, cam.position, cam.view, cam.up, cam.fov, 
					ProjectionParams.centreProj, ProjectionParams.halfVecH, ProjectionParams.halfVecV);

	interceptInfo theRightIntercept = getIntercept (geoms, objectCountInfo, castRay, textureArray);
	glm::vec3 lightVec; 
	for (int i = 0; i < 1; ++ i)
	{
//		lightVec = glm::normalize (multiplyMV (light.transform, glm::vec4 (lightPos.x + (((i%3)-1)*0.5), lightPos.y, lightPos.z + (((i/3)-1)*0.5), 1.0)) - (castRay.origin + (castRay.direction*theRightIntercept.interceptVal)));
		lightVec = glm::normalize (multiplyMV (light.transform, lightPos) - (castRay.origin + (castRay.direction*theRightIntercept.interceptVal)));
		shadedColour += calcShade (theRightIntercept, lightVec, cam.position, castRay, textureArray, ka, ks, kd, lightCol);
	}
//	shadedColour /= nLights;

	// Specular reflection
	// -------------------
	castRay.origin += theRightIntercept.interceptVal*castRay.direction;	// Store the intersection point in castRay.
	castRay.direction = castRay.origin - cam.position;		// We have ray starting at camera and pointing toward intersection point
	castRay.direction = glm::normalize (reflectRay (castRay.direction, theRightIntercept.intrNormal)); // Reflect around intersection normal to compute shade of reflections. 
	
	// Find the intersection point of reflected ray.
	float hasReflective = theRightIntercept.intrMaterial.hasReflective;
	theRightIntercept = getIntercept (geoms, objectCountInfo, castRay, textureArray);
	lightVec = glm::normalize (multiplyMV (light.transform, lightPos) - (castRay.origin + (castRay.direction*theRightIntercept.interceptVal)));
	if (hasReflective)
		shadedColour = ((shadedColour * (float)0.92) + (calcShade (theRightIntercept, lightVec, cam.position, castRay, textureArray, ka, ks, kd, lightCol) * (float)0.08));

	// Shadow shading
	// --------------
//	castRay.origin = castRay.origin + theRightIntercept.interceptVal*castRay.direction;	// Store the intersection point in castRay.
	//castRay.origin += ((float)0.001*theRightIntercept.intrNormal);		// Perturb it along the normal a slight distance to avoid self intersection.
	//
	//glm::vec3 shadowColour = glm::vec3 (0);
	//for (int i = 0; i < nLights; ++ i)
	//{
	//	lightVec = multiplyMV (light.transform, glm::vec4 (lightPos.x + (((i%3)-1)*0.5), lightPos.y, lightPos.z + (((i/3)-1)*0.5), 1.0));
	//	castRay.direction = glm::normalize (lightVec - castRay.origin);

	//	if (isShadowRayBlocked (castRay, lightVec, geoms, objectCountInfo))
	//		shadowColour += (float)0.1 * theRightIntercept.intrMaterial.color;
	//}
	//if ((shadowColour.x != 0) || (shadowColour.y != 0) || (shadowColour.z != 0))
	//	shadedColour = shadowColour/nLights;

	colors [index] = shadedColour;
  }
}

__device__ bool isShadowRayBlocked (ray r, glm::vec3 lightPos, staticGeom *geomsList, sceneInfo objectCountInfo)
{
	float min = 1e6, interceptValue;
	glm::vec3 intrPoint, intrNormal;
	
	for (int i = 0; i < objectCountInfo.nCubes; ++i)
	{
		staticGeom currentGeom = geomsList [i];
		interceptValue = boxIntersectionTest(currentGeom, r, intrPoint, intrNormal);
		if (interceptValue > 0)
		{
			if (interceptValue < min)
				min = interceptValue;
		}
	}

	for (int i = objectCountInfo.nCubes; i <= (objectCountInfo.nCubes+objectCountInfo.nSpheres); ++i)
	{
		staticGeom currentGeom = geomsList [i];
		interceptValue = sphereIntersectionTest(currentGeom, r, intrPoint, intrNormal);
		if (interceptValue > 0)
		{
			if (interceptValue < min)
				min = interceptValue;
		}
	}

//	if (min > 0)
		if (glm::length (lightPos - r.origin) > (min+0.001))
			return true;
	return false;
}

// At each pixel, trace a shadow ray to the light and see if it intersects something else.
__global__ void		shadowFeeler (glm::vec3 startPoint, glm::vec3 lightPosition, glm::vec3 *colorBuffer, staticGeom *geoms, int nGeoms)
{
	;
}

// This function intersects a ray r with all the cubes in the scene and returns the lowest positive intersection value.
//__device__ float intersectRayWithCubes (ray r, staticGeom *cubesList, int nCubes)
//{
//	float min = -0.001;
//	for (int i = 0; i < nCubes; i ++)
//	{
//		staticGeom currentGeom = cubesList [i];
//		
//		interceptValue = boxIntersectionTest(currentGeom, castRay, intrPoint, intrNormal);
//		if (interceptValue < abs (min))
//		{
//			min = interceptValue;
//
//			theRightIntercept.interceptVal = min;
//			theRightIntercept.intrNormal = intrNormal;
//			theRightIntercept.intrMaterial = textureArray [currentGeom.materialid];
//		}
//	}
//}

//// This funcion intersects a ray r with all the spheres in the scene and returns the lowest positive intersection value.
//__device__ float intersectRayWithSpheres (ray r, staticGeom *spheresList, int nSpheres)
//{
//	float min = -0.001;
//	for (int i = 0; i < nCubes; i ++)
//	{	
//		staticGeom currentGeom = cubesList [i];
//
//		interceptValue = sphereIntersectionTest(currentGeom, castRay, intrPoint, intrNormal);
//		if (interceptValue <  abs (min))
//		{
//			min = interceptValue;
//
//			theRightIntercept.interceptVal = min;
//			theRightIntercept.intrNormal = intrNormal;
//			theRightIntercept.intrMaterial = textureArray [currentGeom.materialid];
//		}
//	}
//}

// Kernel for shading cubes.
__global__ void		cubeShade  (glm::vec2 resolution, int nIteration, cameraData camDetails, int rayDepth, 
								glm::vec3 *colorBuffer, staticGeom *cubesList, int nCubes, material *textureData, projectionInfo ProjParams)
{
	;
}

// Kernel for shading spheres.
__global__ void		sphereShade  (glm::vec2 resolution, int nIteration, cameraData camDetails, int rayDepth, 
								glm::vec3 *colorBuffer, staticGeom *spheresList, int nSpheres, material *textureData, projectionInfo ProjParams)
{
	;
}

//TODO: Almost Done!
// Wrapper for the __global__ call that sets up the kernel calls and does a ton of memory management
void cudaRaytraceCore(uchar4* PBOpos, camera* renderCam, int frame, int iterations, material* materials, int numberOfMaterials, geom* geoms, int numberOfGeoms){
  
  int traceDepth = 1; //determines how many bounces the raytracer traces
  projectionInfo	ProjectionParams;
//  setupProjection (ProjectionParams, renderCam->positions [frame], renderCam->ups [frame], renderCam->views [frame], renderCam->fov);
  float degToRad = 3.1415926 / 180.0;
  ProjectionParams.centreProj = renderCam->positions [frame]+renderCam->views [frame];
	glm::vec3	eyeToProjCentre = ProjectionParams.centreProj - renderCam->positions [frame];
	glm::vec3	A = glm::cross (eyeToProjCentre, renderCam->ups [frame]);
	glm::vec3	B = glm::cross (A, eyeToProjCentre);
	float		lenEyeToProjCentre = glm::length (eyeToProjCentre);
	
	ProjectionParams.halfVecH = glm::normalize (A) * lenEyeToProjCentre * (float)tan ((renderCam->fov.x*degToRad) / 2.0);
	ProjectionParams.halfVecV = glm::normalize (B) * lenEyeToProjCentre * (float)tan ((renderCam->fov.y*degToRad) / 2.0);

  // set up crucial magic
  int tileSize = 8;
  dim3 threadsPerBlock(tileSize, tileSize);
  dim3 fullBlocksPerGrid((int)ceil(float(renderCam->resolution.x)/float(tileSize)), (int)ceil(float(renderCam->resolution.y)/float(tileSize)));
  
  //send image to GPU
  glm::vec3* cudaimage = NULL;
  hipMalloc((void**)&cudaimage, (int)renderCam->resolution.x*(int)renderCam->resolution.y*sizeof(glm::vec3));
  hipMemcpy( cudaimage, renderCam->image, (int)renderCam->resolution.x*(int)renderCam->resolution.y*sizeof(glm::vec3), hipMemcpyHostToDevice);
  
  //package geometry and materials and sent to GPU
  staticGeom* geomList = new staticGeom[numberOfGeoms];
  sceneInfo		primCounts;
  
  int count = 1;
  bool lightSet = false;
  for(int i=0; i<numberOfGeoms; i++)
  {
	  if ((geoms [i].materialid == 8) && !lightSet)
	  {
		staticGeom newStaticGeom;
		newStaticGeom.type = geoms[i].type;
		newStaticGeom.materialid = geoms[i].materialid;
		newStaticGeom.translation = geoms[i].translations[frame];
		newStaticGeom.rotation = geoms[i].rotations[frame];
		newStaticGeom.scale = geoms[i].scales[frame];
		newStaticGeom.transform = geoms[i].transforms[frame];
		newStaticGeom.inverseTransform = geoms[i].inverseTransforms[frame];
		geomList[0] = newStaticGeom;
		
		lightSet = true;
	  }

	  else if (geoms [i].type == CUBE)
	  {
		staticGeom newStaticGeom;
		newStaticGeom.type = geoms[i].type;
		newStaticGeom.materialid = geoms[i].materialid;
		newStaticGeom.translation = geoms[i].translations[frame];
		newStaticGeom.rotation = geoms[i].rotations[frame];
		newStaticGeom.scale = geoms[i].scales[frame];
		newStaticGeom.transform = geoms[i].transforms[frame];
		newStaticGeom.inverseTransform = geoms[i].inverseTransforms[frame];
		geomList[count] = newStaticGeom;
		count ++;
	  }
  }

  if (!lightSet)
  {
	  geomList [0] = geomList [count-1];
	  count --;
  }
  primCounts.nCubes = count;
  
  for(int i=0; i<numberOfGeoms; i++)
  {
	  if (geoms [i].type == SPHERE)
	  {
		staticGeom newStaticGeom;
		newStaticGeom.type = geoms[i].type;
		newStaticGeom.materialid = geoms[i].materialid;
		newStaticGeom.translation = geoms[i].translations[frame];
		newStaticGeom.rotation = geoms[i].rotations[frame];
		newStaticGeom.scale = geoms[i].scales[frame];
		newStaticGeom.transform = geoms[i].transforms[frame];
		newStaticGeom.inverseTransform = geoms[i].inverseTransforms[frame];
		geomList[count] = newStaticGeom;
		count ++;
	  }
  }

  primCounts.nSpheres = count - primCounts.nCubes;

  //if (!lightSet)
  //{
		//staticGeom newStaticGeom;
		//newStaticGeom.type = geoms[0].type;
		//newStaticGeom.materialid = geoms[0].materialid;
		//newStaticGeom.translation = geoms[0].translations[frame];
		//newStaticGeom.rotation = geoms[0].rotations[frame];
		//newStaticGeom.scale = geoms[0].scales[frame];
		//newStaticGeom.transform = geoms[0].transforms[frame];
		//newStaticGeom.inverseTransform = geoms[0].inverseTransforms[frame];
		//geomList[0] = newStaticGeom;
  //}

  primCounts.nMeshes = 0;

  staticGeom* cudageoms = NULL;
  hipMalloc((void**)&cudageoms, numberOfGeoms*sizeof(staticGeom));
  hipMemcpy( cudageoms, geomList, numberOfGeoms*sizeof(staticGeom), hipMemcpyHostToDevice);
  
  material		*materialColours = NULL;
  hipError_t returnCode = hipMalloc((void**)&materialColours, numberOfMaterials*sizeof(material));
  if (returnCode != hipSuccess)
  {
	  std::cout << "\nError while trying to send texture data to the GPU!";
	  std::cin.get ();

	  if (cudaimage)
		  hipFree( cudaimage );
	  if (cudageoms)
		  hipFree( cudageoms );
	  if (materialColours)
		  hipFree (materialColours);
	  
	  cudaimage = NULL;
	  cudageoms = NULL;
	  materialColours = NULL;
	  exit (EXIT_FAILURE);
  }
  else
	  hipMemcpy( materialColours, materials, numberOfMaterials*sizeof(material), hipMemcpyHostToDevice);

  //package camera
  cameraData cam;
  cam.resolution = renderCam->resolution;
  cam.position = renderCam->positions[frame];
  cam.view = renderCam->views[frame];
  cam.up = renderCam->ups[frame];
  cam.fov = renderCam->fov;

//  cudaPrintfInit ();
  //kernel launches
  raytraceRay<<<fullBlocksPerGrid, threadsPerBlock>>>(renderCam->resolution, (float)iterations, cam, traceDepth, cudaimage, cudageoms, primCounts, materialColours, ProjectionParams);
  
  sendImageToPBO<<<fullBlocksPerGrid, threadsPerBlock>>>(PBOpos, renderCam->resolution, cudaimage);

  //retrieve image from GPU
  hipMemcpy( renderCam->image, cudaimage, (int)renderCam->resolution.x*(int)renderCam->resolution.y*sizeof(glm::vec3), hipMemcpyDeviceToHost);

 // cudaPrintfDisplay (stdout, true);
 // cudaPrintfEnd ();
  //free up stuff, or else we'll leak memory like a madman
   if (cudaimage)
		hipFree( cudaimage );
   if (cudageoms)
		hipFree( cudageoms );
   if (materialColours)
		hipFree (materialColours);

   cudaimage = NULL;
   cudageoms = NULL;
   materialColours = NULL;

 // make certain the kernel has completed
  hipDeviceSynchronize();
  
  //  hipFree( cudaimage );
//  hipFree( cudageoms );
//  hipFree (materialColours);
  delete geomList;

  checkCUDAError("Kernel failed!");
}
