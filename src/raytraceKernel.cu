#include "hip/hip_runtime.h"
// CIS565 CUDA Raytracer: A parallel raytracer for Patrick Cozzi's CIS565: GPU Computing at the University of Pennsylvania
// Written by Yining Karl Li, Copyright (c) 2012 University of Pennsylvania
// This file includes code from:
//       Rob Farber for CUDA-GL interop, from CUDA Supercomputing For The Masses: http://www.drdobbs.com/architecture-and-design/cuda-supercomputing-for-the-masses-part/222600097
//       Peter Kutz and Yining Karl Li's GPU Pathtracer: http://gpupathtracer.blogspot.com/
//       Yining Karl Li's TAKUA Render, a massively parallel pathtracing renderer: http://www.yiningkarlli.com

#include <stdio.h>
#include <hip/hip_runtime.h>
#include <cmath>
#include "sceneStructs.h"
#include "glm/glm.hpp"
#include "utilities.h"
#include "raytraceKernel.h"
#include "intersections.h"
#include "interactions.h"

#if CUDA_VERSION >= 5000
    #include <hip/hip_vector_types.h>
#else
    #include <cutil_math.h>
#endif

const glm::vec3 bgColour = glm::vec3 (0.55, 0.25, 0);

void checkCUDAError(const char *msg) {
  hipError_t err = hipGetLastError();
  if( hipSuccess != err) {
    fprintf(stderr, "Cuda error: %s: %s.\n", msg, hipGetErrorString( err) ); 
    exit(EXIT_FAILURE); 
  }
} 

//Sets up the projection half vectors.
void	setupProjection (projectionInfo &ProjectionParams, glm::vec3 eye, glm::vec3 view, glm::vec3 up, glm::vec2 fov)
{
	//Set up the projection variables:
	float	degToRad = 3.1415926 / 180.0;
	float	radToDeg = 1.0 / degToRad;

	ProjectionParams.centreProj = eye+view;
	glm::vec3	eyeToProjCentre = ProjectionParams.centreProj - eye;
	glm::vec3	A = glm::cross (ProjectionParams.centreProj, up);
	glm::vec3	B = glm::cross (A, ProjectionParams.centreProj);
	float		lenEyeToProjCentre = glm::length (eyeToProjCentre);
	
	ProjectionParams.halfVecH = glm::normalize (A) * lenEyeToProjCentre * (float)tan ((fov.x*degToRad));
	ProjectionParams.halfVecV = glm::normalize (B) * lenEyeToProjCentre * (float)tan ((fov.y*degToRad));
}

// Reflects the incidentRay around the normal.
__host__ __device__ glm::vec3 reflectRay (glm::vec3 incidentRay, glm::vec3 normal)
{
	glm::vec3 reflectedRay = incidentRay - (2.0f*glm::dot (incidentRay, normal))*normal;
	return reflectedRay;
}

//LOOK: This function demonstrates how to use thrust for random number generation on the GPU!
//Function that generates static.
__host__ __device__ glm::vec3 generateRandomNumberFromThread(glm::vec2 resolution, float time, int x, int y){
  int index = x + (y * resolution.x);
   
  thrust::default_random_engine rng(hash(index*time));
  thrust::uniform_real_distribution<float> u01(0,1);

  return glm::vec3((float) u01(rng), (float) u01(rng), (float) u01(rng));
}

//TODO: IMPLEMENT THIS FUNCTION
//Function that does the initial raycast from the camera
__host__ __device__ ray raycastFromCameraKernel(glm::vec2 resolution, float time, int x, int y, glm::vec3 eye, glm::vec3 view, glm::vec3 up, glm::vec2 fov, glm::vec3 centreProj,
													glm::vec3	halfVecH, glm::vec3 halfVecV)
{
  ray r;
  r.origin = eye;
  r.direction = glm::vec3(0,0,-1);

  float normDeviceX = (float)x / (resolution.x-1);
  float normDeviceY = 1 - ((float)y / (resolution.y-1));

  glm::vec3 P = centreProj + (2*normDeviceX - 1)*halfVecH + (2*normDeviceY - 1)*halfVecV;
  r.direction = glm::normalize (P - r.origin);

  return r;
}

//Kernel that blacks out a given image buffer
__global__ void clearImage(glm::vec2 resolution, glm::vec3* image){
    int x = (blockIdx.x * blockDim.x) + threadIdx.x;
    int y = (blockIdx.y * blockDim.y) + threadIdx.y;
    int index = x + (y * resolution.x);
    if(x<=resolution.x && y<=resolution.y){
      image[index] = glm::vec3(0,0,0);
    }
}

//Kernel that writes the image to the OpenGL PBO directly.
__global__ void sendImageToPBO(uchar4* PBOpos, glm::vec2 resolution, glm::vec3* image, int nLights){
  
  int x = (blockIdx.x * blockDim.x) + threadIdx.x;
  int y = (blockIdx.y * blockDim.y) + threadIdx.y;
  int index = x + (y * resolution.x);
  
  if(x<=resolution.x && y<=resolution.y){
	  image [index] /= nLights;
      glm::vec3 color;
      color.x = image[index].x*255.0;
      color.y = image[index].y*255.0;
      color.z = image[index].z*255.0;

      if(color.x>255){
        color.x = 255;
      }

      if(color.y>255){
        color.y = 255;
      }

      if(color.z>255){
        color.z = 255;
      }
      
      // Each thread writes one pixel location in the texture (textel)
      PBOpos[index].w = 0;
      PBOpos[index].x = color.x;
      PBOpos[index].y = color.y;
      PBOpos[index].z = color.z;
  }
}

__device__ interceptInfo getIntercept (staticGeom * geoms, sceneInfo objectCountInfo, ray castRay, material* textureArray)
{
	glm::vec3 intrPoint = glm::vec3 (0, 0, 0);
	glm::vec3 intrNormal = glm::vec3 (0, 0, 0);

	float interceptValue = -32767;

	material newMaterial;
	newMaterial.color = glm::vec3 (0,0,0);
	newMaterial.specularExponent = 1.0;
	newMaterial.hasReflective = 0.0;
	newMaterial.hasRefractive = 0.0;

	interceptInfo theRightIntercept;					// Stores the lowest intercept.
	theRightIntercept.interceptVal = interceptValue;	// Initially, it is empty/invalid
	theRightIntercept.intrNormal = intrNormal;			// Intially, Normal - 0,0,0
	theRightIntercept.intrMaterial = newMaterial;

	float min = 1e6;
	// Two different loops to intersect ray with cubes and spheres.
	for (int i = 0; i < objectCountInfo.nCubes; ++i)
	{
		staticGeom currentGeom = geoms [i];

		interceptValue = boxIntersectionTest(currentGeom, castRay, intrPoint, intrNormal);
		if (interceptValue > 0)
		{
			if (interceptValue < min)
			{
				min = interceptValue;

				theRightIntercept.interceptVal = min;
				theRightIntercept.intrNormal = intrNormal;
				theRightIntercept.intrMaterial = textureArray [currentGeom.materialid];
			}
		}
	}

	for (int i = objectCountInfo.nCubes; i <= (objectCountInfo.nCubes+objectCountInfo.nSpheres); ++i)
	{
		staticGeom currentGeom = geoms [i];

		interceptValue = sphereIntersectionTest(currentGeom, castRay, intrPoint, intrNormal);
		if (interceptValue > 0)
		{
			if (interceptValue < min)
			{
				min = interceptValue;

				theRightIntercept.interceptVal = min;
				theRightIntercept.intrNormal = intrNormal;
				theRightIntercept.intrMaterial = textureArray [currentGeom.materialid];
			}
		}
	}

	return theRightIntercept;
}

__device__ glm::vec3 calcShade (interceptInfo theRightIntercept, glm::vec3 lightVec, glm::vec3 eye, ray castRay, material* textureArray, float ka, float ks, float kd, glm::vec3 lightCol)
{
	glm::vec3 shadedColour = glm::vec3 (0,0,0);
	if (theRightIntercept.interceptVal > 0)
	{
		// Ambient shading
		shadedColour = ka * theRightIntercept.intrMaterial.color;

		// Diffuse shading
		glm::vec3 intrPoint = castRay.origin + theRightIntercept.interceptVal*castRay.direction;	// The intersection point.
		glm::vec3 intrNormal = glm::normalize (eye - intrPoint); // intrNormal is the view vector.
		float interceptValue = max (glm::dot (theRightIntercept.intrNormal, lightVec), (float)0); // Diffuse Lighting is given by (N.L); N being normal at intersection pt and L being light vector.
		intrPoint = (theRightIntercept.intrMaterial.color * kd * interceptValue);			// Reuse intrPoint to store partial product (kdId) of the diffuse shading computation.
		shadedColour += multiplyVV (lightCol, intrPoint);		// shadedColour will have diffuse shaded colour. 

		// Specular shading
		lightVec = glm::normalize (reflectRay (-lightVec, theRightIntercept.intrNormal)); // Reuse lightVec for storing the reflection of light vector around the normal.
		interceptValue = max (glm::dot (lightVec, intrNormal), (float)0);				// Reuse interceptValue for computing dot pdt of specular.
		shadedColour += (lightCol * ks * pow (interceptValue, theRightIntercept.intrMaterial.specularExponent));
	}
	
	return	shadedColour;
}

//TODO: Done!
//Core raytracer kernel
__global__ void raytraceRay(glm::vec2 resolution, float time, cameraData cam, int rayDepth, glm::vec3* colors,
                            staticGeom* geoms, sceneInfo objectCountInfo, material* textureArray, projectionInfo ProjectionParams,
							glm::vec3 lightPosition)
{
  __shared__ staticGeom light;
  __shared__ float ks;
  __shared__ float ka;
  __shared__ float kd;
  __shared__ glm::vec3 lightPos;
  __shared__ glm::vec3 lightCol;
  __shared__ float nLights;
  __shared__ int sqrtLights;
  __shared__ float stepSize;

  if ((threadIdx.x == 0) && (threadIdx.y == 0))
  {
	  ks = 0.5;
	  ka = 0.1;
	  kd = 1-ks-ka;
	  nLights = 64;
	  sqrtLights = sqrt (nLights);
	  stepSize = 1.0/(sqrtLights-1);
	  light = geoms [0];
	  lightPos = /*multiplyMV (light.transform, */lightPosition/*)*/;
	  lightCol = (textureArray [light.materialid].color /** textureArray [light.materialid].emittance*/);
  }
  __syncthreads ();

  int x = (blockIdx.x * blockDim.x) + threadIdx.x;
  int y = (blockIdx.y * blockDim.y) + threadIdx.y;
  int index = x + (y * resolution.x);

  glm::vec3 shadedColour = glm::vec3 (0);

  if((x<=resolution.x && y<=resolution.y))
  {
    ray castRay = raycastFromCameraKernel(resolution, time, x, y, cam.position, cam.view, cam.up, cam.fov, 
					ProjectionParams.centreProj, ProjectionParams.halfVecH, ProjectionParams.halfVecV);

	interceptInfo theRightIntercept = getIntercept (geoms, objectCountInfo, castRay, textureArray);
	glm::vec3 lightVec; 
//	for (int i = 0; i < nLights; ++ i)
//	{
//		glm::vec3 tmpLightPos = multiplyMV (light.transform, lightPosition/*glm::vec3 (lightPos.x+ ((i%sqrtLights)*stepSize), lightPos.y, lightPos.z + ((i/sqrtLights)*stepSize)*/));
		lightVec = glm::normalize (lightPosition - (castRay.origin + (castRay.direction*theRightIntercept.interceptVal)));
		shadedColour += calcShade (theRightIntercept, lightVec, cam.position, castRay, textureArray, ka, ks, kd, lightCol);
//	}

//	shadedColour /= nLights;
	glm::vec3 rightnormal = theRightIntercept.intrNormal;

	// Specular reflection
	// -------------------
	castRay.origin += theRightIntercept.interceptVal*castRay.direction;	// Store the intersection point in castRay.
	castRay.direction = castRay.origin - cam.position;		// We have ray starting at camera and pointing toward intersection point
	castRay.direction = glm::normalize (reflectRay (castRay.direction, theRightIntercept.intrNormal)); // Reflect around intersection normal to compute shade of reflections. 
	
	// Find the intersection point of reflected ray and calculate shade there.
	float hasReflective = theRightIntercept.intrMaterial.hasReflective;
	theRightIntercept = getIntercept (geoms, objectCountInfo, castRay, textureArray);
	// Use only a point light to calculate the shade of reflection, since it doesn't matter much anyway.
	lightVec = glm::normalize (lightPosition - (castRay.origin + (castRay.direction*theRightIntercept.interceptVal)));
	if (hasReflective)
		shadedColour = ((shadedColour * (float)0.92) + (calcShade (theRightIntercept, lightVec, cam.position, castRay, textureArray, ka, ks, kd, lightCol) * (float)0.08));

//	 Shadow shading
//	 --------------
	castRay.origin += ((float)0.04*rightnormal);		// Perturb the intersection pt along the normal a slight distance 
														// to avoid self intersection.
	glm::vec3 shadowColour = glm::vec3 (0);
//	for (int i = 0; i < nLights; ++ i)
//	{
//		lightVec = multiplyMV (light.transform, glm::vec4 (lightPos.x + ((i%sqrtLights)*stepSize), lightPos.y, lightPos.z + ((i/sqrtLights)*stepSize), 1.0));
		castRay.direction = glm::normalize (lightPosition - castRay.origin);

		if (isShadowRayBlocked (castRay, lightPosition, geoms, objectCountInfo))
			/*shadowColour +=*/shadedColour = ka * theRightIntercept.intrMaterial.color;	// If point in shadow, add ambient colour to shadowColour
//		else
//			shadowColour += shadedColour;								// Otherwise, add the computed shade.
//	}
//	shadedColour = shadowColour/nLights;

	colors [index] += shadedColour;
  }
}

__device__ bool isShadowRayBlocked (ray r, glm::vec3 lightPos, staticGeom *geomsList, sceneInfo objectCountInfo)
{
	float min = 1e6, interceptValue;
	glm::vec3 intrPoint, intrNormal;
	
	for (int i = 0; i < objectCountInfo.nCubes; ++i)
	{
		staticGeom currentGeom = geomsList [i];
		interceptValue = boxIntersectionTest(currentGeom, r, intrPoint, intrNormal);
		if (interceptValue > 0)
		{
			if (interceptValue < min)
				min = interceptValue;
		}
	}

	for (int i = objectCountInfo.nCubes; i <= (objectCountInfo.nCubes+objectCountInfo.nSpheres); ++i)
	{
		staticGeom currentGeom = geomsList [i];
		interceptValue = sphereIntersectionTest(currentGeom, r, intrPoint, intrNormal);
		if (interceptValue > 0)
		{
			if (interceptValue < min)
				min = interceptValue;
		}
	}

//	if (min > 0)
		if (glm::length (lightPos - r.origin) > (min+0.1))
			return true;
	return false;
}

// At each pixel, trace a shadow ray to the light and see if it intersects something else.
__global__ void		shadowFeeler (glm::vec2 resolution, float time, cameraData cam, int rayDepth, glm::vec3* colors,
                            staticGeom* geoms, sceneInfo objectCountInfo, material* textureArray, projectionInfo ProjectionParams, 
							renderInfo* renderParams)
{
	__shared__ staticGeom light;
	__shared__ float ks;
	__shared__ float ka;
	__shared__ float kd;
	__shared__ glm::vec3 lightPos;
	__shared__ glm::vec3 lightCol;
	__shared__ float nLights;
	__shared__ int sqrtLights;
	__shared__ float stepSize;

	if ((threadIdx.x == 0) && (threadIdx.y == 0))
	{
		ks = renderParams->ks;
		ka = renderParams->ka;
		kd = renderParams->kd;
		nLights = renderParams->nLights;
		sqrtLights = renderParams->sqrtLights;
		stepSize = renderParams->lightStepSize;
		light = geoms [0];
		lightPos = renderParams->lightPos;
		lightCol = renderParams->lightCol;
	}
	__syncthreads ();

	int x = (blockIdx.x * blockDim.x) + threadIdx.x;
	int y = (blockIdx.y * blockDim.y) + threadIdx.y;
    int index = x + (y * resolution.x);
	
	if ((x <= resolution.x) && (y <= resolution.y)) 
	{
		ray castRay = raycastFromCameraKernel(resolution, time, x, y, cam.position, cam.view, cam.up, cam.fov, 
					ProjectionParams.centreProj, ProjectionParams.halfVecH, ProjectionParams.halfVecV);

		interceptInfo theRightIntercept = getIntercept (geoms, objectCountInfo, castRay, textureArray);
		glm::vec3 lightVec; 
	
	//	Shadow shading
	//	--------------
		// Perturb the intersection pt along the normal a slight distance to avoid self intersection. 
		castRay.origin += (castRay.direction * (float)(theRightIntercept.interceptVal - 0.001));
															
		glm::vec3 shadedColour = colors [index];
		glm::vec3 shadowColour = glm::vec3 (0);
		for (int i = 0; i < nLights; ++ i)
		{
			lightVec = multiplyMV (light.transform, glm::vec4 (lightPos.x + ((i%sqrtLights)*stepSize), lightPos.y, lightPos.z + ((i/sqrtLights)*stepSize), 1.0));
			castRay.direction = glm::normalize (lightVec - castRay.origin);

			if (isShadowRayBlocked (castRay, lightVec, geoms, objectCountInfo))
				shadowColour += ka * theRightIntercept.intrMaterial.color;	// If point in shadow, add ambient colour to shadowColour
			else
				shadowColour += shadedColour;								// Otherwise, add the computed shade.
		}
		shadedColour = shadowColour/nLights;

		colors [index] = shadedColour;
	}
}

// Kernel for shading cubes.
__global__ void		cubeShade  (glm::vec2 resolution, int nIteration, cameraData camDetails, int rayDepth, 
								glm::vec3 *colorBuffer, staticGeom *cubesList, int nCubes, material *textureData, projectionInfo ProjParams)
{
	;
}

// Kernel for shading spheres.
__global__ void		sphereShade  (glm::vec2 resolution, int nIteration, cameraData camDetails, int rayDepth, 
								glm::vec3 *colorBuffer, staticGeom *spheresList, int nSpheres, material *textureData, projectionInfo ProjParams)
{
	;
}

//TODO: Done!
// Wrapper for the __global__ call that sets up the kernel calls and does a ton of memory management
void cudaRaytraceCore(uchar4* PBOpos, camera* renderCam, int frame, int iterations, material* materials, int numberOfMaterials, geom* geoms, int numberOfGeoms){
  
  int traceDepth = 1; //determines how many bounces the raytracer traces
  projectionInfo	ProjectionParams;
//  setupProjection (ProjectionParams, renderCam->positions [frame], renderCam->ups [frame], renderCam->views [frame], renderCam->fov);
  float degToRad = 3.1415926 / 180.0;
  ProjectionParams.centreProj = renderCam->positions [frame]+renderCam->views [frame];
	glm::vec3	eyeToProjCentre = ProjectionParams.centreProj - renderCam->positions [frame];
	glm::vec3	A = glm::cross (eyeToProjCentre, renderCam->ups [frame]);
	glm::vec3	B = glm::cross (A, eyeToProjCentre);
	float		lenEyeToProjCentre = glm::length (eyeToProjCentre);
	
	ProjectionParams.halfVecH = glm::normalize (A) * lenEyeToProjCentre * (float)tan ((renderCam->fov.x*degToRad) / 2.0);
	ProjectionParams.halfVecV = glm::normalize (B) * lenEyeToProjCentre * (float)tan ((renderCam->fov.y*degToRad) / 2.0);

  // set up crucial magic
  int tileSize = 8;
  dim3 threadsPerBlock(tileSize, tileSize);
  dim3 fullBlocksPerGrid((int)ceil(float(renderCam->resolution.x)/float(tileSize)), (int)ceil(float(renderCam->resolution.y)/float(tileSize)));
  
  //send image to GPU
  glm::vec3* cudaimage = NULL;
  hipMalloc((void**)&cudaimage, (int)renderCam->resolution.x*(int)renderCam->resolution.y*sizeof(glm::vec3));
  hipMemcpy( cudaimage, renderCam->image, (int)renderCam->resolution.x*(int)renderCam->resolution.y*sizeof(glm::vec3), hipMemcpyHostToDevice);
  
  // Package render information such as no. of point lights to use to approximate area light and diffuse, specular and ambient coeffs
  renderInfo *renderParamsDeviceCopy = NULL;
  hipMalloc((void**)&renderParamsDeviceCopy, sizeof(renderInfo));  
  renderInfo renderParams;
  renderParams.ks = 0.3;
  renderParams.ka = 0.1;
  renderParams.kd = 1-renderParams.ks-renderParams.ka;
  renderParams.nLights = 64;
  renderParams.sqrtLights = sqrt ((float)renderParams.nLights);
  renderParams.lightStepSize = 1.0/(renderParams.sqrtLights-1);
  hipMemcpy (renderParamsDeviceCopy, &renderParams, sizeof(renderInfo), hipMemcpyHostToDevice);

  //package geometry and materials and sent to GPU
  staticGeom* geomList = new staticGeom[numberOfGeoms];
  sceneInfo		primCounts;
  
  int count = 1;	int lightIndex = 0;
  bool lightSet = false;
  for(int i=0; i<numberOfGeoms; i++)
  {
	  if ((geoms [i].materialid == 8) && !lightSet)
	  {
		staticGeom newStaticGeom;
		newStaticGeom.type = geoms[i].type;
		newStaticGeom.materialid = geoms[i].materialid;
		newStaticGeom.translation = geoms[i].translations[frame];
		newStaticGeom.rotation = geoms[i].rotations[frame];
		newStaticGeom.scale = geoms[i].scales[frame];
		newStaticGeom.transform = geoms[i].transforms[frame];
		newStaticGeom.inverseTransform = geoms[i].inverseTransforms[frame];
		geomList[0] = newStaticGeom;
		
		lightSet = true;
		lightIndex = i;
	  }

	  else if (geoms [i].type == CUBE)
	  {
		staticGeom newStaticGeom;
		newStaticGeom.type = geoms[i].type;
		newStaticGeom.materialid = geoms[i].materialid;
		newStaticGeom.translation = geoms[i].translations[frame];
		newStaticGeom.rotation = geoms[i].rotations[frame];
		newStaticGeom.scale = geoms[i].scales[frame];
		newStaticGeom.transform = geoms[i].transforms[frame];
		newStaticGeom.inverseTransform = geoms[i].inverseTransforms[frame];
		geomList[count] = newStaticGeom;
		count ++;
	  }
  }

  if (!lightSet)
  {
	  geomList [0] = geomList [count-1];
	  count --;
  }
  primCounts.nCubes = count;
  
  for(int i=0; i<numberOfGeoms; i++)
  {
	  if (geoms [i].type == SPHERE)
	  {
		staticGeom newStaticGeom;
		newStaticGeom.type = geoms[i].type;
		newStaticGeom.materialid = geoms[i].materialid;
		newStaticGeom.translation = geoms[i].translations[frame];
		newStaticGeom.rotation = geoms[i].rotations[frame];
		newStaticGeom.scale = geoms[i].scales[frame];
		newStaticGeom.transform = geoms[i].transforms[frame];
		newStaticGeom.inverseTransform = geoms[i].inverseTransforms[frame];
		geomList[count] = newStaticGeom;
		count ++;
	  }
  }

  primCounts.nSpheres = count - primCounts.nCubes;

  //if (!lightSet)
  //{
		//staticGeom newStaticGeom;
		//newStaticGeom.type = geoms[0].type;
		//newStaticGeom.materialid = geoms[0].materialid;
		//newStaticGeom.translation = geoms[0].translations[frame];
		//newStaticGeom.rotation = geoms[0].rotations[frame];
		//newStaticGeom.scale = geoms[0].scales[frame];
		//newStaticGeom.transform = geoms[0].transforms[frame];
		//newStaticGeom.inverseTransform = geoms[0].inverseTransforms[frame];
		//geomList[0] = newStaticGeom;
  //}

  primCounts.nMeshes = 0;

  staticGeom* cudageoms = NULL;
  hipMalloc((void**)&cudageoms, numberOfGeoms*sizeof(staticGeom));
  hipMemcpy( cudageoms, geomList, numberOfGeoms*sizeof(staticGeom), hipMemcpyHostToDevice);
  
  material		*materialColours = NULL;
  hipError_t returnCode = hipMalloc((void**)&materialColours, numberOfMaterials*sizeof(material));
  if (returnCode != hipSuccess)
  {
	  std::cout << "\nError while trying to send texture data to the GPU!";
	  std::cin.get ();

	  if (cudaimage)
		  hipFree( cudaimage );
	  if (cudageoms)
		  hipFree( cudageoms );
	  if (materialColours)
		  hipFree (materialColours);
	  
	  cudaimage = NULL;
	  cudageoms = NULL;
	  materialColours = NULL;
	  exit (EXIT_FAILURE);
  }
  else
	  hipMemcpy( materialColours, materials, numberOfMaterials*sizeof(material), hipMemcpyHostToDevice);

  renderInfo	RenderParams;
  RenderParams.kd;
  RenderParams.ka;
  RenderParams.ks;
  RenderParams.nLights = 64;
  RenderParams.sqrtLights = sqrt ((float)RenderParams.nLights);
  RenderParams.lightStepSize = 1.0/(RenderParams.sqrtLights-1);
  RenderParams.lightPos = glm::vec3 (-0.5, -0.6, -0.5);
  RenderParams.lightCol = materials [geoms [lightIndex].materialid].color;

  //package camera
  cameraData cam;
  cam.resolution = renderCam->resolution;
  cam.position = renderCam->positions[frame];
  cam.view = renderCam->views[frame];
  cam.up = renderCam->ups[frame];
  cam.fov = renderCam->fov;

  glm::vec3 lightPos = glm::vec3 (-0.5, -0.6, -0.5);
  for (int i = 0; i < RenderParams.nLights; i ++)
  {
	  lightPos = multiplyMV (geomList [0].transform, glm::vec4 (RenderParams.lightPos.x + ((i%RenderParams.sqrtLights)*RenderParams.lightStepSize), 
				RenderParams.lightPos.y, RenderParams.lightPos.z + ((i/RenderParams.sqrtLights)*RenderParams.lightStepSize), 1.0));
	  
	  // kernel launches
	  raytraceRay<<<fullBlocksPerGrid, threadsPerBlock>>>(renderCam->resolution, (float)iterations, cam, traceDepth, cudaimage, cudageoms, primCounts, materialColours, ProjectionParams, lightPos);
	  hipDeviceSynchronize();
	  std::cout << "\rRendering.. " <<  ceil ((float)i/(RenderParams.nLights-1) * 100) << " percent complete.";
  }
  sendImageToPBO<<<fullBlocksPerGrid, threadsPerBlock>>>(PBOpos, renderCam->resolution, cudaimage, RenderParams.nLights);
  std::cout << "\n";
  //retrieve image from GPU
  hipMemcpy( renderCam->image, cudaimage, (int)renderCam->resolution.x*(int)renderCam->resolution.y*sizeof(glm::vec3), hipMemcpyDeviceToHost);

 // cudaPrintfDisplay (stdout, true);
 // cudaPrintfEnd ();
  //free up stuff, or else we'll leak memory like a madman
   if (cudaimage)
		hipFree( cudaimage );
   if (cudageoms)
		hipFree( cudageoms );
   if (materialColours)
		hipFree (materialColours);

   cudaimage = NULL;
   cudageoms = NULL;
   materialColours = NULL;

 // make certain the kernel has completed
  hipDeviceSynchronize();
  
  //  hipFree( cudaimage );
//  hipFree( cudageoms );
//  hipFree (materialColours);
  delete geomList;

  checkCUDAError("Kernel failed!");
}
