#include "hip/hip_runtime.h"
// CIS565 CUDA Raytracer: A parallel raytracer for Patrick Cozzi's CIS565: GPU Computing at the University of Pennsylvania
// Written by Yining Karl Li, Copyright (c) 2012 University of Pennsylvania
// This file includes code from:
//       Rob Farber for CUDA-GL interop, from CUDA Supercomputing For The Masses: http://www.drdobbs.com/architecture-and-design/cuda-supercomputing-for-the-masses-part/222600097
//       Peter Kutz and Yining Karl Li's GPU Pathtracer: http://gpupathtracer.blogspot.com/
//       Yining Karl Li's TAKUA Render, a massively parallel pathtracing renderer: http://www.yiningkarlli.com

#include <thrust/device_vector.h>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <cmath>
#include "sceneStructs.h"
#include "glm/glm.hpp"
#include "utilities.h"
#include "raytraceKernel.h"
#include "intersections.h"
#include "interactions.h"
//#include <vector>

#if CUDA_VERSION >= 5000
    #include <hip/hip_vector_types.h>
#else
    #include <cutil_math.h>
#endif

const glm::vec3 bgColour = glm::vec3 (0.55, 0.25, 0);

void checkCUDAError(const char *msg) {
  hipError_t err = hipGetLastError();
  if( hipSuccess != err) {
    fprintf(stderr, "Cuda error: %s: %s.\n", msg, hipGetErrorString( err) ); 
    exit(EXIT_FAILURE); 
  }
} 

//Sets up the projection half vectors.
void	setupProjection (projectionInfo &ProjectionParams, glm::vec3 eye, glm::vec3 view, glm::vec3 up, glm::vec2 fov)
{
	//Set up the projection variables:
	float	degToRad = 3.1415926 / 180.0;
	float	radToDeg = 1.0 / degToRad;

	ProjectionParams.centreProj = eye+view;
	glm::vec3	eyeToProjCentre = ProjectionParams.centreProj - eye;
	glm::vec3	A = glm::cross (ProjectionParams.centreProj, up);
	glm::vec3	B = glm::cross (A, ProjectionParams.centreProj);
	float		lenEyeToProjCentre = glm::length (eyeToProjCentre);
	
	ProjectionParams.halfVecH = glm::normalize (A) * lenEyeToProjCentre * (float)tan ((fov.x*degToRad));
	ProjectionParams.halfVecV = glm::normalize (B) * lenEyeToProjCentre * (float)tan ((fov.y*degToRad));
}

// Reflects the incidentRay around the normal.
__host__ __device__ glm::vec3 reflectRay (glm::vec3 incidentRay, glm::vec3 normal)
{
	glm::vec3 reflectedRay = incidentRay - (2.0f*glm::dot (incidentRay, normal))*normal;
	return reflectedRay;
}

//LOOK: This function demonstrates how to use thrust for random number generation on the GPU!
//Function that generates static.
__host__ __device__ glm::vec3 generateRandomNumberFromThread(glm::vec2 resolution, float time, int x, int y){
  int index = x + (y * resolution.x);
   
  thrust::default_random_engine rng(hash(index*time));
  thrust::uniform_real_distribution<float> u01(0,1);

  return glm::vec3((float) u01(rng), (float) u01(rng), (float) u01(rng));
}

//TODO: IMPLEMENT THIS FUNCTION
//Function that does the initial raycast from the camera
__host__ __device__ ray raycastFromCameraKernel(glm::vec2 resolution, float time, int x, int y, glm::vec3 eye, glm::vec3 view, glm::vec3 up, glm::vec2 fov, glm::vec3 centreProj,
													glm::vec3	halfVecH, glm::vec3 halfVecV)
{
  ray r;
  r.origin = eye;
  r.direction = glm::vec3(0,0,-1);

 // float	degToRad = 3.1415926 / 180.0;
 // float	radToDeg = 1.0 / degToRad;

	//ProjectionParams.centreProj = eye+view;
	//glm::vec3	eyeToProjCentre = ProjectionParams.centreProj - eye;
	//glm::vec3	A = glm::cross (ProjectionParams.centreProj, up);
	//glm::vec3	B = glm::cross (A, ProjectionParams.centreProj);
	//float		lenEyeToProjCentre = glm::length (eyeToProjCentre);
	//
	//ProjectionParams.halfVecH = glm::normalize (A) * lenEyeToProjCentre * (float)tan ((fov.x*degToRad) / 2.0);
	//ProjectionParams.halfVecV = glm::normalize (B) * lenEyeToProjCentre * (float)tan ((fov.y*degToRad) / 2.0);


  float normDeviceX = (float)x / (resolution.x-1);
  float normDeviceY = 1 - ((float)y / (resolution.y-1));

  glm::vec3 P = /*ProjectionParams.*/centreProj + (2*normDeviceX - 1)*/*ProjectionParams.*/halfVecH + (2*normDeviceY - 1)*/*ProjectionParams.*/halfVecV;
  r.direction = glm::normalize (P - r.origin);

  return r;
}

//Kernel that blacks out a given image buffer
__global__ void clearImage(glm::vec2 resolution, glm::vec3* image){
    int x = (blockIdx.x * blockDim.x) + threadIdx.x;
    int y = (blockIdx.y * blockDim.y) + threadIdx.y;
    int index = x + (y * resolution.x);
    if(x<=resolution.x && y<=resolution.y){
      image[index] = glm::vec3(0,0,0);
    }
}

//Kernel that writes the image to the OpenGL PBO directly.
__global__ void sendImageToPBO(uchar4* PBOpos, glm::vec2 resolution, glm::vec3* image){
  
  int x = (blockIdx.x * blockDim.x) + threadIdx.x;
  int y = (blockIdx.y * blockDim.y) + threadIdx.y;
  int index = x + (y * resolution.x);
  
  if(x<=resolution.x && y<=resolution.y){

      glm::vec3 color;
      color.x = image[index].x*255.0;
      color.y = image[index].y*255.0;
      color.z = image[index].z*255.0;

      if(color.x>255){
        color.x = 255;
      }

      if(color.y>255){
        color.y = 255;
      }

      if(color.z>255){
        color.z = 255;
      }
      
      // Each thread writes one pixel location in the texture (textel)
      PBOpos[index].w = 0;
      PBOpos[index].x = color.x;
      PBOpos[index].y = color.y;
      PBOpos[index].z = color.z;
  }
}

//TODO: IMPLEMENT THIS FUNCTION
//Core raytracer kernel
__global__ void raytraceRay(glm::vec2 resolution, float time, cameraData cam, int rayDepth, glm::vec3* colors,
                            staticGeom* geoms, int numberOfGeoms, material* textureArray, projectionInfo ProjectionParams){

  int x = (blockIdx.x * blockDim.x) + threadIdx.x;
  int y = (blockIdx.y * blockDim.y) + threadIdx.y;
  int index = x + (y * resolution.x);
  staticGeom light;
  bool lightSet = false;

  float ks = 0.3, ka = 0.2, kd = 1-ks-ka, specEx = 30.0;

  if((x<=resolution.x && y<=resolution.y))
  {
	  ray castRay = raycastFromCameraKernel(resolution, time, x, y, cam.position, cam.view, cam.up, cam.fov, 
					ProjectionParams.centreProj, ProjectionParams.halfVecH, ProjectionParams.halfVecV);
//	
	glm::vec3 zeroVector = glm::vec3 (0, 0, 0);
	glm::vec3 intrPoint = glm::vec3 (0, 0, 0);
	glm::vec3 intrNormal = glm::vec3 (0, 0, 0);

	float interceptValue = -32767;

	interceptInfo theRightIntercept;					// Stores the lowest intercept.
	theRightIntercept.interceptVal = interceptValue;			// Initially, it is empty/invalid
	theRightIntercept.intrNormal = intrNormal;		// Normal - 0,0,0
	theRightIntercept.intrMaterial = intrPoint;		// Colour - black;

	float min = 1e6;
	for (int i = 0; i < numberOfGeoms; ++i)
	{
		if (geoms [i].type == SPHERE)
		{	
			interceptValue = sphereIntersectionTest(geoms [i], castRay, intrPoint, intrNormal);
			if (interceptValue > 0)
			{
				if (interceptValue < min)
				{
					min = interceptValue;

					theRightIntercept.interceptVal = min;
					theRightIntercept.intrNormal = intrNormal;
					theRightIntercept.intrMaterial = textureArray [geoms [i].materialid].color;
				}
			}
		}
		else if (geoms [i].type == CUBE)
		{	
			interceptValue = boxIntersectionTest(geoms [i], castRay, intrPoint, intrNormal);
			if (interceptValue > 0)
			{
				if (interceptValue < min)
				{
					min = interceptValue;

					theRightIntercept.interceptVal = min;
					theRightIntercept.intrNormal = intrNormal;
					theRightIntercept.intrMaterial = textureArray [geoms [i].materialid].color;
				}
			}
		}

		if (geoms [i].materialid == 8)
		{	
			light = geoms [i];
			lightSet = true;
		}
	}

	if ((lightSet) && (theRightIntercept.interceptVal > 0))
	{
		glm::vec3 lightPos = glm::vec3 (0, -0.5, 0);
		lightPos = multiplyMV (light.transform, glm::vec4 (lightPos.x, lightPos.y, lightPos.z, 1.0));

		// Ambient shading
		colors [index] = glm::vec3 (ka * theRightIntercept.intrMaterial);

		glm::vec3 surfDiffuseColour;
		glm::vec3 lightVec = glm::normalize (lightPos - intrPoint);
		
		// Diffuse shading
		intrPoint = castRay.origin + theRightIntercept.interceptVal*castRay.direction;
		float dotPdt = max (glm::dot (theRightIntercept.intrNormal, lightVec), (float)0);
		surfDiffuseColour = (theRightIntercept.intrMaterial * kd * dotPdt);
		colors [index] += multiplyVV (textureArray [light.materialid].color, surfDiffuseColour);

		// Specular shading
		glm::vec3 viewVec = cam.position - intrPoint;
		glm::vec3 reflLightVec = reflectRay (-lightVec, theRightIntercept.intrNormal);
		float specularDotPdt = max (glm::dot (reflLightVec, viewVec), (float)0);
		colors [index] += (textureArray [light.materialid].color * ks * pow (specularDotPdt, textureArray [light.materialid].specularExponent));

//		colors [index] += multiplyVV (textureArray [light->materialid] * diffuseSpecColour);
	}
	else
	{
		colors[index] = zeroVector;
	}
  }
}

//TODO: FINISH THIS FUNCTION
// Wrapper for the __global__ call that sets up the kernel calls and does a ton of memory management
void cudaRaytraceCore(uchar4* PBOpos, camera* renderCam, int frame, int iterations, material* materials, int numberOfMaterials, geom* geoms, int numberOfGeoms){
  
  int traceDepth = 1; //determines how many bounces the raytracer traces
  projectionInfo	ProjectionParams;
//  setupProjection (ProjectionParams, renderCam->positions [frame], renderCam->ups [frame], renderCam->views [frame], renderCam->fov);
  float degToRad = 3.1415926 / 180.0;
  ProjectionParams.centreProj = renderCam->positions [frame]+renderCam->views [frame];
	glm::vec3	eyeToProjCentre = ProjectionParams.centreProj - renderCam->positions [frame];
	glm::vec3	A = glm::cross (eyeToProjCentre, renderCam->ups [frame]);
	glm::vec3	B = glm::cross (A, eyeToProjCentre);
	float		lenEyeToProjCentre = glm::length (eyeToProjCentre);
	
	ProjectionParams.halfVecH = glm::normalize (A) * lenEyeToProjCentre * (float)tan ((renderCam->fov.x*degToRad) / 2.0);
	ProjectionParams.halfVecV = glm::normalize (B) * lenEyeToProjCentre * (float)tan ((renderCam->fov.y*degToRad) / 2.0);

  // set up crucial magic
  int tileSize = 8;
  dim3 threadsPerBlock(tileSize, tileSize);
  dim3 fullBlocksPerGrid((int)ceil(float(renderCam->resolution.x)/float(tileSize)), (int)ceil(float(renderCam->resolution.y)/float(tileSize)));
  
  //send image to GPU
  glm::vec3* cudaimage = NULL;
  hipMalloc((void**)&cudaimage, (int)renderCam->resolution.x*(int)renderCam->resolution.y*sizeof(glm::vec3));
  hipMemcpy( cudaimage, renderCam->image, (int)renderCam->resolution.x*(int)renderCam->resolution.y*sizeof(glm::vec3), hipMemcpyHostToDevice);
  
  //package geometry and materials and sent to GPU
  staticGeom* geomList = new staticGeom[numberOfGeoms];
  for(int i=0; i<numberOfGeoms; i++){
    staticGeom newStaticGeom;
    newStaticGeom.type = geoms[i].type;
    newStaticGeom.materialid = geoms[i].materialid;
    newStaticGeom.translation = geoms[i].translations[frame];
    newStaticGeom.rotation = geoms[i].rotations[frame];
    newStaticGeom.scale = geoms[i].scales[frame];
    newStaticGeom.transform = geoms[i].transforms[frame];
    newStaticGeom.inverseTransform = geoms[i].inverseTransforms[frame];
    geomList[i] = newStaticGeom;
  }
  
  staticGeom* cudageoms = NULL;
  hipMalloc((void**)&cudageoms, numberOfGeoms*sizeof(staticGeom));
  hipMemcpy( cudageoms, geomList, numberOfGeoms*sizeof(staticGeom), hipMemcpyHostToDevice);
  
  material		*materialColours = NULL;
  hipError_t returnCode = hipMalloc((void**)&materialColours, numberOfMaterials*sizeof(material));
  if (returnCode != hipSuccess)
  {
	  std::cout << "\nError while trying to send texture data to the GPU!";
	  std::cin.get ();

	  if (cudaimage)
		  hipFree( cudaimage );
	  if (cudageoms)
		  hipFree( cudageoms );
	  if (materialColours)
		  hipFree (materialColours);
	  
	  cudaimage = NULL;
	  cudageoms = NULL;
	  materialColours = NULL;
	  exit (EXIT_FAILURE);
  }
  else
	  hipMemcpy( materialColours, materials, numberOfMaterials*sizeof(material), hipMemcpyHostToDevice);

  //package camera
  cameraData cam;
  cam.resolution = renderCam->resolution;
  cam.position = renderCam->positions[frame];
  cam.view = renderCam->views[frame];
  cam.up = renderCam->ups[frame];
  cam.fov = renderCam->fov;

//  cudaPrintfInit ();
  //kernel launches
  raytraceRay<<<fullBlocksPerGrid, threadsPerBlock>>>(renderCam->resolution, (float)iterations, cam, traceDepth, cudaimage, cudageoms, numberOfGeoms, materialColours, ProjectionParams);
  
  sendImageToPBO<<<fullBlocksPerGrid, threadsPerBlock>>>(PBOpos, renderCam->resolution, cudaimage);

  //retrieve image from GPU
  hipMemcpy( renderCam->image, cudaimage, (int)renderCam->resolution.x*(int)renderCam->resolution.y*sizeof(glm::vec3), hipMemcpyDeviceToHost);

 // cudaPrintfDisplay (stdout, true);
 // cudaPrintfEnd ();
  //free up stuff, or else we'll leak memory like a madman
   if (cudaimage)
		hipFree( cudaimage );
   if (cudageoms)
		hipFree( cudageoms );
   if (materialColours)
		hipFree (materialColours);

   cudaimage = NULL;
   cudageoms = NULL;
   materialColours = NULL;

 // make certain the kernel has completed
  hipDeviceSynchronize();
  
  //  hipFree( cudaimage );
//  hipFree( cudageoms );
//  hipFree (materialColours);
  delete geomList;

  checkCUDAError("Kernel failed!");
}
